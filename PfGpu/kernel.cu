
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>

#include "CSVReader.h"
#include "time_stamp.h"


int main()
{
	//Load Data
	CSVReader rawdatafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5UwbData.data.csv");
	
	CSVReader resultdatafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5RealPose.csv");
	CSVReader beacondatafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5beaconset.data.csv");
	//CSVReader datafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5UwbData.data.csv");
	//std::cout << "test:kkkk:" << std::endl;
	std::cout << "raw uwb data:" << rawdatafile.GetMatrix().GetRows()<<":"
		<< rawdatafile.GetMatrix().GetCols() << std::endl;

	std::cout << "result data :" << resultdatafile.GetMatrix().GetRows()<<":"
		<< resultdatafile.GetMatrix().GetCols() << std::endl;

	std::cout << "beaconset :" << beacondatafile.GetMatrix().GetRows()<<":"
		<< beacondatafile.GetMatrix().GetCols() << std::endl;

	int beacon_num = rawdatafile.GetMatrix().GetCols() - 1;
	int raw_num_times = rawdatafile.GetMatrix().GetRows();
	int dimisional = 3;
	double z_offset = 1.12;

	double *h_raw = new double[(beacon_num + 1)*(raw_num_times + 1)];
	double *h_beaconset = new double[(beacon_num + 1)*dimisional];
	double *h_result = new double[(dimisional)*(raw_num_times + 1)];

	for(int i(0);i<raw_num_times;i++)
	{
		for(int j(0);j<beacon_num;++j)
		{
			h_raw[i, j] = *rawdatafile.GetMatrix()(i, j + 1);
		}
	}

	for(int i(0);i<beacon_num;++i)
	{
		for(int j(0);j<dimisional;++j)
		{
			h_beaconset[i, j] = *beacondatafile.GetMatrix()(i, j);
		}
	}

	for(int i(0);i<raw_num_times;++i)
	{
		for(int j(0);j<dimisional;++j)
		{
			h_result[i, j] = *resultdatafile.GetMatrix()(i, j);
		}
	}

	//Save to array
	double *d_raw;
	double *d_beaconset;
	hipMalloc((void**)&d_raw, (beacon_num + 1)*(raw_num_times + 1) * sizeof(double));
	hipMalloc((void**)&d_beaconset, (beacon_num + 1)*(dimisional) * sizeof(double));


	//PF
	int particle_num = 10000;
	double *p_state = NULL;
	double *p_score = NULL;

	hipMalloc((void**)&p_state, particle_num*dimisional * sizeof(double));
	hipMalloc((void **)&p_score, particle_num * 1 * sizeof(double));



	//Sample

	//Evaluate

	//Get result


	//Resample

	//Compute error


	//save output.

	int a(0);
	std::cin >> a;
	return 0;

}

