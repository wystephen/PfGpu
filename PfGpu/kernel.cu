#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <stdio.h>

#include <iostream>

#include "CSVReader.h"
#include "time_stamp.h"

#include "UwbParticleFilter.cuh"
#include <hiprand/hiprand_kernel.h>
#pragma comment(lib,"cudart.lib")
#pragma comment(lib,"hiprand.lib")

int main()
{
	//Load Data
	CSVReader rawdatafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5UwbData.data.csv");
	
	CSVReader resultdatafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5RealPose.csv");
	CSVReader beacondatafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5beaconset.data.csv");
	//CSVReader datafile("C:\\Users\\steve\\Documents\\Visual\ Studio\ 2015\\Projects\\PfGpu\\Debug\\5UwbData.data.csv");
	//std::cout << "test:kkkk:" << std::endl;
	std::cout << "raw uwb data:" << rawdatafile.GetMatrix().GetRows()<<":"
		<< rawdatafile.GetMatrix().GetCols() << std::endl;

	std::cout << "result data :" << resultdatafile.GetMatrix().GetRows()<<":"
		<< resultdatafile.GetMatrix().GetCols() << std::endl;

	std::cout << "beaconset :" << beacondatafile.GetMatrix().GetRows()<<":"
		<< beacondatafile.GetMatrix().GetCols() << std::endl;

	int beacon_num = rawdatafile.GetMatrix().GetCols() - 1;
	int raw_num_times = rawdatafile.GetMatrix().GetRows();
	int dimisional = 3;
	float z_offset = 1.12;

	float *h_raw = new float[(beacon_num )*(raw_num_times)];
	float *h_beaconset = new float[(beacon_num + 1)*dimisional];
	float *h_result = new float[(dimisional)*(raw_num_times )];

	for(int i(0);i<raw_num_times;i++)
	{
		for(int j(0);j<beacon_num;++j)
		{
			h_raw[i, j] = *rawdatafile.GetMatrix()(i, j + 1);
		}
	}

	for(int i(0);i<beacon_num;++i)
	{
		for(int j(0);j<dimisional;++j)
		{
			h_beaconset[i, j] = *beacondatafile.GetMatrix()(i, j);
		}
	}

	for(int i(0);i<raw_num_times;++i)
	{
		for(int j(0);j<dimisional;++j)
		{
			h_result[i, j] = *resultdatafile.GetMatrix()(i, j);
		}
	}

	//Save to array
	float *d_raw;
	float *d_beaconset;
	hipMalloc((void**)&d_raw, (beacon_num )*(raw_num_times ) * sizeof(float));
	hipMalloc((void**)&d_beaconset, (beacon_num )*(dimisional) * sizeof(float));
	
	hipMemcpy(d_raw, h_raw, (beacon_num)*(raw_num_times)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_beaconset, h_beaconset, (beacon_num )*(dimisional) * sizeof(float), hipMemcpyHostToDevice);


	//PF
	int particle_num = 10000;
	double sample_sigma = 0.5;

	int thread_dim_x = 100;


	float *p_state = NULL;
	float *p_score = NULL;
	float *rnd_x = NULL;
	float *rnd_y = NULL;

	hipMalloc((void**)&p_state, particle_num*dimisional * sizeof(float));
	hipMalloc((void **)&p_score, particle_num * 1 * sizeof(float));
	hipMalloc((void **)&rnd_x, particle_num * 1 * sizeof(float));
	hipMalloc((void **)&rnd_y, particle_num * 1 * sizeof(float));

	//Random generator 
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, long long(TimeStamp::now()*100));

	//INITIAL
	//-9.9777 - 6.0391	1.4184
	InitialParticle <<<ceil(particle_num/thread_dim_x),thread_dim_x >>> (p_state, p_score,
		-9.977, -6.0391, 1.4184,
		particle_num);
	


	//Sample
	hiprandGenerateNormal(gen, rnd_x, particle_num , 0.0, sample_sigma);
	hiprandGenerateNormal(gen, rnd_y, particle_num , 0.0, sample_sigma);
	Sample <<<ceil(particle_num / thread_dim_x), thread_dim_x >>> (p_state, rnd_x, rnd_y,
		1, particle_num);

	//Evaluate
	//COMMIT


	//Get result


	//Resample

	//Compute error


	//save output.
	float *h_p_state = new float[particle_num*dimisional];
	float *h_p_score = new float[particle_num];
	hipMemcpy(h_p_state,p_state, particle_num*dimisional * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_p_score,p_score, particle_num * sizeof(float), hipMemcpyDeviceToHost);
	for(int i(0);i<particle_num;++i)
	{
		for(int j(0);j<dimisional;++j)
		{
			std::cout << h_p_state[i*dimisional + j] << "-";
		}
		std::cout << std::endl;
		std::cout << h_p_score[i] << "   =====" << std::endl;
	}

	int a(0);
	std::cin >> a;

	hipFree(d_raw);
	hipFree(d_beaconset);

	return 0;

}

